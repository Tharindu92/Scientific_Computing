#include "hip/hip_runtime.h"
// Source: http://web.mit.edu/pocky/www/cudaworkshop/MonteCarlo/Pi.cu

// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu  December 22, 2010.
//Derived somewhat from code developed by Patrick Rogers, UNC-C

#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <getopt.h>
#include <omp.h>

#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256
#define PI 3.1415926535  // known value of pi

/*
 * \Run CuRand
 */
//run on GPU
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	float x, y;

	hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


	for(int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}

__global__ void gpu_monte_carlo_d(double *estimate, hiprandState *states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	double x, y;

	hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


	for(int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (double) TRIALS_PER_THREAD; // return estimate of pi
}

//run on CPU without threads
float host_monte_carlo(long trials) {
	float x, y;
	long points_in_circle = 0;
	for(long i = 0; i < trials; i++) {
		x = rand() / (float) RAND_MAX;
		y = rand() / (float) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

double host_monte_carlo_d(long trials) {
	double x, y;
	long points_in_circle = 0;
	for(long i = 0; i < trials; i++) {
		x = rand() / (double) RAND_MAX;
		y = rand() / (double) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

//run on cpu with threads
float para_monte_carlo(long trials, int thread_count) {
	float x, y;
	long points_in_circle = 0;
	#pragma omp parallel num_threads(thread_count) private(x,y)
	{
		#pragma omp for schedule(static) reduction(+:points_in_circle)
		for(long i = 0; i < trials; i++) {
			x = rand() / (float) RAND_MAX;
			y = rand() / (float) RAND_MAX;
			points_in_circle += (x*x + y*y <= 1.0f);
		}
	}

	return 4.0f * points_in_circle / trials;
}

double para_monte_carlo_d(long trials, int thread_count) {
	double x, y;
	long points_in_circle = 0;
	#pragma omp parallel num_threads(thread_count) private(x,y)
	{
		#pragma omp for schedule(static) reduction(+:points_in_circle)
		for(long i = 0; i < trials; i++) {
			x = rand() / (double) RAND_MAX;
			y = rand() / (double) RAND_MAX;
			points_in_circle += (x*x + y*y <= 1.0f);
		}
	}
	return 4.0f * points_in_circle / trials;
}

/*
 * pi-hiprand-thrust
 */

struct estimate_pi :
    public thrust::unary_function<unsigned int, float>
{
  __device__
  float operator()(unsigned int thread_id)
  {
    float sum = 0;
    unsigned int N = 8192; // samples per thread

    unsigned int seed = thread_id;

    hiprandState s;

    // seed a random number generator
    hiprand_init(seed, 0, 0, &s);

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      // draw a sample from the unit square
      float x = hiprand_uniform(&s);
      float y = hiprand_uniform(&s);

      // measure distance from the origin
      float dist = sqrtf(x*x + y*y);

      // add 1.0f if (u0,u1) is inside the quarter circle
      if(dist <= 1.0f)
        sum += 1.0f;
    }

    // multiply by 4 to get the area of the whole circle
    sum *= 4.0f;

    // divide by N
    return sum / N;
  }
};

struct estimate_pi_d :
    public thrust::unary_function<unsigned int, double>
{
  __device__
  double operator()(unsigned int thread_id)
  {
    double sum = 0;
    unsigned int N = 8192; // samples per thread

    unsigned int seed = thread_id;

    hiprandState s;

    // seed a random number generator
    hiprand_init(seed, 0, 0, &s);

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      // draw a sample from the unit square
      double x = hiprand_uniform(&s);
      double y = hiprand_uniform(&s);

      // measure distance from the origin
      double dist = sqrtf(x*x + y*y);

      // add 1.0f if (u0,u1) is inside the quarter circle
      if(dist <= 1.0f)
        sum += 1.0f;
    }

    // multiply by 4 to get the area of the whole circle
    sum *= 4.0f;

    // divide by N
    return sum / N;
  }
};

int main(int argc, char **argv) {
	int dp = 0;
	int c;
	int thread_count = 2;
	while((c = getopt(argc, argv, "dn:")) != -1){
		switch(c){
			case 'd':
				dp = 1;
				break;
			case 'n':
				thread_count = atoi(optarg);
				if(thread_count > 8 || thread_count < 2){
					printf("Invalid Number of threads\nThread number is set to 2\n");
					thread_count = 2;
				}
				break;
			default:
				dp = 0;
				break;
		}
	}
	clock_t start, stop;
	int M = 32768;
	if(!dp){
		printf("Run with single precision\n");
		float host[BLOCKS * THREADS];
		float *dev;
		hiprandState *devStates;

		printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD,
	BLOCKS, THREADS);

		start = clock();

		hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(float)); // allocate device mem. for counts

		hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );

		gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);

		hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost); // return results

		float pi_gpu;
		for(int i = 0; i < BLOCKS * THREADS; i++) {
			pi_gpu += host[i];
		}

		pi_gpu /= (BLOCKS * THREADS);

		stop = clock();

		printf("GPU CuRand pi calculated in %lf s.\n", (double)(stop-start)/CLOCKS_PER_SEC);

		start = clock();
		float pi_cpu = host_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD);
		stop = clock();
		printf("CPU pi calculated in %lf s.\n", (double)(stop-start)/CLOCKS_PER_SEC);

		start = clock();
		float pi_para = para_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD, thread_count);
		stop = clock();
		printf("CPU with %d threads pi calculated in %lf s.\n", thread_count,(double)(stop-start)/CLOCKS_PER_SEC);

		start = clock();
		float estimate = thrust::transform_reduce(
			          thrust::counting_iterator<int>(0),
			          thrust::counting_iterator<int>(M),
			          estimate_pi(),
			          0.0f,
			          thrust::plus<float>());
			    estimate /= M;
		stop = clock();
		printf("CUDA Thrust CuRand pi calculated in %lf s.\n",(double)(stop-start)/CLOCKS_PER_SEC);

		printf("CUDA CuRand estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
		printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI);
		printf("CPU with %d threads estimate of PI = %f [error of %f]\n", thread_count, pi_para, pi_para - PI);
		printf("CUDA Thrust CuRand estimate of PI = %f [error of %f]\n", estimate, estimate - PI);

	}else{
		printf("Run with double precision\n");
		double host[BLOCKS * THREADS];
		double *dev;
		hiprandState *devStates;

		printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD,
	BLOCKS, THREADS);

		start = clock();

		hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(double)); // allocate device mem. for counts

		hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );

		gpu_monte_carlo_d<<<BLOCKS, THREADS>>>(dev, devStates);

		hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(double), hipMemcpyDeviceToHost); // return results

		double pi_gpu;
		for(int i = 0; i < BLOCKS * THREADS; i++) {
			pi_gpu += host[i];
		}

		pi_gpu /= (BLOCKS * THREADS);

		stop = clock();

		printf("GPU CuRand pi calculated in %lf s.\n", (stop-start)/(double)CLOCKS_PER_SEC);

		start = clock();
		double pi_cpu = host_monte_carlo_d(BLOCKS * THREADS * TRIALS_PER_THREAD);
		stop = clock();
		printf("CPU pi calculated in %lf s.\n", (stop-start)/(double)CLOCKS_PER_SEC);

		start = clock();
		double pi_para = para_monte_carlo_d(BLOCKS * THREADS * TRIALS_PER_THREAD, thread_count);
		stop = clock();
		printf("CPU with %d threads pi calculated in %lf s.\n", thread_count,(double)(stop-start)/CLOCKS_PER_SEC);

		start = clock();
		double estimate = thrust::transform_reduce(
			          thrust::counting_iterator<int>(0),
			          thrust::counting_iterator<int>(M),
			          estimate_pi_d(),
			          0.0f,
			          thrust::plus<double>());
			    estimate /= M;
		stop = clock();
		printf("CUDA Thrust CuRand pi calculated in %lf s.\n",(double)(stop-start)/CLOCKS_PER_SEC);


		printf("CUDA CuRand estimate of PI = %lf [error of %lf]\n", pi_gpu, pi_gpu - PI);
		printf("CPU estimate of PI = %lf [error of %lf]\n", pi_cpu, pi_cpu - PI);
		printf("CPU with %d threads estimate of PI = %lf [error of %lf]\n", thread_count, pi_para, pi_para - PI);
		printf("CUDA Thrust CuRand estimate of PI = %lf [error of %lf]\n", estimate, estimate - PI);
	}

	return 0;
}

